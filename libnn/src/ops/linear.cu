#include "hip/hip_runtime.h"
#include <libnn/ops/linear.h>

__global__
void cu_linear_forward(size_t M, const float* weights, const float* inp, float* out)
{
	size_t n = blockIdx.x * blockDim.x + threadIdx.x;
	out[n] = 0;
	for (size_t i = 0; i < M; i++)
		out[n] += weights[n * M + i] * inp[i];
}

void nn::linear_impl::forward(size_t M, size_t N, const float* weights, const float* inp, float* out)
{
	// bad implementation for now, will improve it later with less malloc/frees
	float* cu_weights;
	float* cu_inp;
	float* cu_out;
	hipMalloc(&cu_weights, M * N * sizeof(float));
	hipMalloc(&cu_inp, M * sizeof(float));
	hipMalloc(&cu_out, N * sizeof(float));
	hipMemcpy(cu_weights, weights, M * N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(cu_inp, inp, M * sizeof(float), hipMemcpyHostToDevice);

	int blocknum = 256;
	cu_linear_forward<<<(N + blocknum - 1) / blocknum, blocknum >>>(M, cu_weights, cu_inp, cu_out);

	hipMemcpy(out, cu_out, N * sizeof(float), hipMemcpyDeviceToHost);

	hipFree(cu_weights);
	hipFree(cu_inp);
	hipFree(cu_out);
}
